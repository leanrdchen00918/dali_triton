#include "myoperation.h"
#include <iostream>

namespace other_ns {
  void MyOp::Run(::dali::MixedWorkspace &ws) {

    auto &input = ws.Input<::dali::CPUBackend>(0);
    int num_samples = input.shape().num_samples();
    auto &output = ws.Output<::dali::GPUBackend>(0);
    output.set_type<size_t>();
    size_t tmp_size[2] = {output.capacity(), input.capacity()};
    output.Resize(::dali::uniform_list_shape(num_samples, std::vector<int64_t>{2}));
    for (int sample_idx = 0; sample_idx < num_samples; sample_idx++) {
      auto *out = output.mutable_tensor<size_t>(sample_idx);
      CUDA_CALL(hipStreamSynchronize(ws.stream()));
      CUDA_CALL(hipMemcpy(out, &tmp_size, sizeof(size_t) * 2, hipMemcpyHostToDevice));
    }
    std::cout << "MyOp::Run is done." << std::endl;
  }
}  // namespace other_ns


DALI_REGISTER_OPERATOR(MyOperation, ::other_ns::MyOp, ::dali::Mixed);
DALI_SCHEMA(MyOperation)
  .DocStr("MyOp")
  .NumInput(1)
  .NumOutput(1);

